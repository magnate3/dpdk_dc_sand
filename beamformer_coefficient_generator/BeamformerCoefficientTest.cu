#include "hip/hip_runtime.h"
#include "BeamformerCoefficientTest.hpp"
#include "Utils.hpp"
#include <hip/hip_complex.h>
#include <cmath>
#include <iostream>

float ts_diff(struct timespec first, struct timespec last)
// Give the difference between two timespecs, in floats. For opencl calculations.
{
    float time_difference = (float) last.tv_sec - (float) first.tv_sec;
    long nanosec_difference = last.tv_nsec - first.tv_nsec;
    time_difference += (float) nanosec_difference / 1e9; //Should work if this is negative as well?
    return time_difference;
}

BeamformerCoeffTest::BeamformerCoeffTest(float fFloatingPointTolerance):
    m_fFloatingPointTolerance(fFloatingPointTolerance),
    m_ulSizeSteeringCoefficients(NR_STATIONS * NR_BEAMS * sizeof(hipFloatComplex)),
    m_ulSizeDelayValues(NR_STATIONS * NR_BEAMS * sizeof(struct delay_vals_extended))
{   
    //Get timestamp of now
    struct timespec m_sCurrentTime_ns;
    clock_gettime(CLOCK_MONOTONIC, &m_sCurrentTime_ns);
    
    //Initialising Memory
    GPU_ERRCHK(hipHostMalloc((void**)&m_pHDelayValues,m_ulSizeDelayValues));
    GPU_ERRCHK(hipMalloc((void**)&m_pDDelayValues,m_ulSizeDelayValues));

    GPU_ERRCHK(hipHostMalloc((void**)&m_pfHSteeringCoeffs,m_ulSizeSteeringCoefficients));
    GPU_ERRCHK(hipMalloc((void**)&m_pfDSteeringCoeffs,m_ulSizeSteeringCoefficients));

    //Generating Block Sizes
    int numSamplesPerChannel = NR_STATIONS*NR_BEAMS;
    int numBlocksPerChannel = numSamplesPerChannel/NUM_THREADS_PER_BLOCK;
    int threadsPerBlock = 0;
    if(numSamplesPerChannel%NUM_THREADS_PER_BLOCK != 0){
        numBlocksPerChannel++;
    }
    if(numBlocksPerChannel > 1){
        threadsPerBlock = NUM_THREADS_PER_BLOCK;
    }else{
        threadsPerBlock = numSamplesPerChannel;
    }
    m_cudaGridSize = dim3(numBlocksPerChannel,NR_STATIONS);
    m_cudaBlockSize = dim3(threadsPerBlock);
}

BeamformerCoeffTest::~BeamformerCoeffTest()
{
    GPU_ERRCHK(hipFree(m_pDDelayValues));
    GPU_ERRCHK(hipHostFree(m_pHDelayValues));
    GPU_ERRCHK(hipFree(m_pfDSteeringCoeffs));
    GPU_ERRCHK(hipHostFree(m_pfHSteeringCoeffs));
}

void BeamformerCoeffTest::simulate_input()
{
    float fSamplingPeriod = 1e-9;
    struct timespec sRefTime;
    sRefTime.tv_sec = m_sCurrentTime_ns.tv_sec;
    sRefTime.tv_nsec = m_sCurrentTime_ns.tv_nsec;

    #define TIME_SHIFT  50000
    if (sRefTime.tv_nsec >= TIME_SHIFT)
        sRefTime.tv_nsec -= TIME_SHIFT;
    else
    {
        sRefTime.tv_sec -= 1;
        sRefTime.tv_nsec += (1000000000 - TIME_SHIFT);
    }

    size_t ulNumDelayVelays = NR_STATIONS*NR_BEAMS;
    for (size_t i = 0; i < NR_STATIONS*NR_BEAMS; i++)
    {
        m_pHDelayValues[i].fSamplingPeriod_s = fSamplingPeriod;
        m_pHDelayValues[i].sRefTime_ns = sRefTime;
        m_pHDelayValues[i].fDelay_s = ((float)i/ulNumDelayVelays)*fSamplingPeriod/3; //let's make them in a linear ramp
        m_pHDelayValues[i].fDelayRate_sps = 2e-11;
        m_pHDelayValues[i].fPhase_rad = (1 -((float)i/ulNumDelayVelays))*fSamplingPeriod/3;
        m_pHDelayValues[i].fPhaseRate_radps = 3e-11;
    }
}

void BeamformerCoeffTest::transfer_HtoD()
{

}

void BeamformerCoeffTest::run_kernel()
{

}

void BeamformerCoeffTest::transfer_DtoH()
{

}

void BeamformerCoeffTest::verify_output()
{
    for (int c = 0; c < NR_CHANNELS; c++)
    {
        for (int a = 0; a < NR_STATIONS; a++)
        {
            for (int b = 0; b < NR_BEAMS; b++)
            {   
                //Generate simulated data
                struct delay_vals_extended sDelayVal = m_pHDelayValues[a*NR_BEAMS + b];
                float fDeltaTime = ts_diff(sDelayVal.sRefTime_ns, m_sCurrentTime_ns);
                float fDeltaDelay = sDelayVal.fDelayRate_sps*fDeltaTime;
                float fDelayN = (sDelayVal.fDelayRate_sps + fDeltaDelay)*c*M_PI/(sDelayVal.fSamplingPeriod_s*NR_CHANNELS);
                float fDelayN2 = (sDelayVal.fDelay_s + fDeltaDelay)*(NR_CHANNELS/2)*M_PI/(sDelayVal.fSamplingPeriod_s*NR_CHANNELS);
                float fDeltaPhase = sDelayVal.fPhaseRate_radps*fDeltaTime;
                float fPhase0 = sDelayVal.fPhase_rad - fDelayN2 + fDeltaPhase;
                float fRotation = fDelayN + fPhase0;
                float fSteeringCoeffCorrectReal = cos(fRotation);//At least i think its the real one - may need to check this if its important
                float fSteeringCoeffCorrectImag = sin(fRotation);

                //Get data generated on GPU
                size_t ulCoeffIndex =  2*(c*NR_STATIONS*NR_BEAMS + a*NR_BEAMS + b);
                float fSteeringCoeffGeneratedReal = m_pfHSteeringCoeffs[ulCoeffIndex];
                float fSteeringCoeffGeneratedImag = m_pfHSteeringCoeffs[ulCoeffIndex+1];

                if(std::abs(fSteeringCoeffGeneratedReal - fSteeringCoeffCorrectReal) > m_fFloatingPointTolerance
                    || std::abs(fSteeringCoeffGeneratedImag - fSteeringCoeffCorrectImag) > m_fFloatingPointTolerance)
                {
                    //std::cout << fSteeringCoeffGeneratedReal << " " <<  fSteeringCoeffCorrectReal << std::endl;
                    std::cout << fSteeringCoeffGeneratedImag << " " <<  fSteeringCoeffCorrectImag << std::endl;
                    m_iResult = -1;
                    return;
                }

                //cplx_beamweights[2*(c*n_antennas*n_beams + a*n_beams + b)] = cos(rotation);
                //cplx_beamweights[2*(c*n_antennas*n_beams + a*n_beams + b)+1] = sin(rotation);
                //std::cout << " c: " << c << " b: " << b << " a: " << a << " r: " << cplx_beamweights[2*(c*n_antennas*n_beams + a*n_beams + b)] << " i: " << cplx_beamweights[2*(c*n_antennas*n_beams + a*n_beams + b)+1] <<std::endl; 
            }
        }
    }
    m_iResult = 1;
}