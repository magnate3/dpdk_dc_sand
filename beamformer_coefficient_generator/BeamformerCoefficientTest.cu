#include "hip/hip_runtime.h"
#include <hip/hip_complex.h>
#include <cmath>
#include <iostream>

#include "BeamformerCoefficientTest.hpp"
#include "BeamformerKernels.cuh"
#include "Utils.hpp"
//#include "Kernels.cu"

float ts_diff(struct timespec first, struct timespec last)
// Give the difference between two timespecs, in floats. For opencl calculations.
{
    float time_difference = (float) last.tv_sec - (float) first.tv_sec;
    long nanosec_difference = last.tv_nsec - first.tv_nsec;
    time_difference += (float) nanosec_difference / 1e9f; //Should work if this is negative as well?
    return time_difference;
}

BeamformerCoeffTest::BeamformerCoeffTest(float fFloatingPointTolerance):
    m_fFloatingPointTolerance(fFloatingPointTolerance),
    m_ulSizeSteeringCoefficients(NR_CHANNELS * NR_STATIONS * NR_BEAMS * sizeof(hipFloatComplex)),
    m_ulSizeDelayValues(NR_STATIONS * NR_BEAMS * sizeof(struct delay_vals_extended))
{   
    //Get timestamp of now
    struct timespec m_sCurrentTime_ns;
    clock_gettime(CLOCK_MONOTONIC, &m_sCurrentTime_ns);
    
    //Initialising Memory
    GPU_ERRCHK(hipHostMalloc((void**)&m_pHDelayValues,m_ulSizeDelayValues));
    GPU_ERRCHK(hipMalloc((void**)&m_pDDelayValues,m_ulSizeDelayValues));

    GPU_ERRCHK(hipHostMalloc((void**)&m_pfHSteeringCoeffs,m_ulSizeSteeringCoefficients));
    GPU_ERRCHK(hipMalloc((void**)&m_pfDSteeringCoeffs,m_ulSizeSteeringCoefficients));

    //Generating Block Sizes
    int numSamplesPerChannel = NR_STATIONS*NR_BEAMS;
    int numBlocksPerChannel = numSamplesPerChannel/NUM_THREADS_PER_BLOCK;
    int threadsPerBlock = 0;
    if(numSamplesPerChannel%NUM_THREADS_PER_BLOCK != 0){
        numBlocksPerChannel++;
    }
    if(numBlocksPerChannel > 1){
        threadsPerBlock = NUM_THREADS_PER_BLOCK;
    }else{
        threadsPerBlock = numSamplesPerChannel;
    }
    m_cudaGridSize = dim3(numBlocksPerChannel,NR_STATIONS);//dim3(7,1);//
    m_cudaBlockSize = dim3(threadsPerBlock);
    std::cout << "Block Size: " << threadsPerBlock << std::endl;
    std::cout << "Grid Size: x: " << numBlocksPerChannel << " y: " << NR_STATIONS << std::endl; 

}

BeamformerCoeffTest::~BeamformerCoeffTest()
{
    GPU_ERRCHK(hipFree(m_pDDelayValues));
    GPU_ERRCHK(hipHostFree(m_pHDelayValues));
    GPU_ERRCHK(hipFree(m_pfDSteeringCoeffs));
    GPU_ERRCHK(hipHostFree(m_pfHSteeringCoeffs));
}

void BeamformerCoeffTest::simulate_input()
{
    float fSamplingPeriod = 1e-9f;
    struct timespec sRefTime;
    sRefTime.tv_sec = m_sCurrentTime_ns.tv_sec;
    sRefTime.tv_nsec = m_sCurrentTime_ns.tv_nsec;

    #define TIME_SHIFT  50000
    if (sRefTime.tv_nsec >= TIME_SHIFT)
        sRefTime.tv_nsec -= TIME_SHIFT;
    else
    {
        sRefTime.tv_sec -= 1;
        sRefTime.tv_nsec += (1000000000 - TIME_SHIFT);
    }

    size_t ulNumDelayVelays = NR_STATIONS*NR_BEAMS;
    for (size_t i = 0; i < NR_STATIONS*NR_BEAMS; i++)
    {
        m_pHDelayValues[i].fSamplingPeriod_s = fSamplingPeriod;
        m_pHDelayValues[i].sRefTime_ns = sRefTime;
        m_pHDelayValues[i].fDelay_s = ((float)i/ulNumDelayVelays)*fSamplingPeriod/3; //let's make them in a linear ramp
        m_pHDelayValues[i].fDelayRate_sps = 2e-11;
        m_pHDelayValues[i].fPhase_rad = (1 -((float)i/ulNumDelayVelays))*fSamplingPeriod/3;
        m_pHDelayValues[i].fPhaseRate_radps = 3e-11;
    }
}

void BeamformerCoeffTest::transfer_HtoD()
{
    GPU_ERRCHK(hipMemcpy(m_pDDelayValues,m_pHDelayValues,m_ulSizeDelayValues,hipMemcpyHostToDevice));
}

void BeamformerCoeffTest::run_kernel()
{
    calculate_beamweights_naive<<<m_cudaGridSize,m_cudaBlockSize>>>(m_sCurrentTime_ns,m_pHDelayValues,m_pfDSteeringCoeffs);
}

void BeamformerCoeffTest::transfer_DtoH()
{
    GPU_ERRCHK(hipMemcpy(m_pfHSteeringCoeffs,m_pfDSteeringCoeffs,m_ulSizeSteeringCoefficients,hipMemcpyDeviceToHost));
}

void BeamformerCoeffTest::verify_output()
{
    int temp = 0;

    float * fCorrectDate = (float*)malloc(NR_BEAMS*NR_CHANNELS*NR_STATIONS*2*sizeof(float));
    for (size_t c = 0; c < NR_CHANNELS; c++)
    {
        for (size_t a = 0; a < NR_STATIONS; a++)
        {
            for (size_t b = 0; b < NR_BEAMS; b++)
            {   
                //Generate simulated data
                struct delay_vals_extended sDelayVal = m_pHDelayValues[a*NR_BEAMS + b];
                float fDeltaTime = ts_diff(sDelayVal.sRefTime_ns, m_sCurrentTime_ns);
                float fDeltaDelay = sDelayVal.fDelayRate_sps*fDeltaTime;
                float fDelayN = (sDelayVal.fDelayRate_sps + fDeltaDelay)*c*M_PI/(sDelayVal.fSamplingPeriod_s*NR_CHANNELS);
                float fDelayN2 = (sDelayVal.fDelay_s + fDeltaDelay)*(NR_CHANNELS/2)*M_PI/(sDelayVal.fSamplingPeriod_s*NR_CHANNELS);
                float fDeltaPhase = sDelayVal.fPhaseRate_radps*fDeltaTime;
                float fPhase0 = sDelayVal.fPhase_rad - fDelayN2 + fDeltaPhase;
                float fRotation = fDelayN + fPhase0;
                float fSteeringCoeffCorrectReal = cos(fRotation);//At least i think its the real one - may need to check this if its important
                float fSteeringCoeffCorrectImag = sin(fRotation);

                //Get data generated on GPU
                size_t ulCoeffIndex =  2*(c*NR_STATIONS*NR_BEAMS + a*NR_BEAMS + b);
                fCorrectDate[ulCoeffIndex] = fSteeringCoeffCorrectReal;
                fCorrectDate[ulCoeffIndex+1] = fSteeringCoeffCorrectImag;

                //if(ulCoeffIndex == 1680 /*|| (ulCoeffIndex > 1600 && ulCoeffIndex < 1700)*/ ){
                //    std::cout << ulCoeffIndex << " C: " << c << ", A: " << a << ", B: " << b << " Correct data: " << fSteeringCoeffCorrectReal << " + " << fSteeringCoeffCorrectImag << "j" << std::endl;
                //}
            }
        }
    }

    //std::cout << NR_STATIONS*NR_CHANNELS*NR_BEAMS*2 << std::endl;
    for (size_t i = 0; i < NR_STATIONS*NR_CHANNELS*NR_BEAMS*2; i++)
    {
        if(std::abs(m_pfHSteeringCoeffs[i] - fCorrectDate[i]) > m_fFloatingPointTolerance){
            std::cout << i << " Generated " << m_pfHSteeringCoeffs[i] << " Correct " << fCorrectDate[i] << std::endl;
            // temp++;
            // if(temp == 1){
            m_iResult = -1;
            return;
            // }
        }
    }

    free(fCorrectDate);
    
    m_iResult = 1;
}