
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#define SAMPLES_PER_BLOCK 4096 //Limited by the amount of shared memory available per block
#define N 4096
#define Fs 1712e6
#define Fir_length 256

__global__ void kernel_ddc(float *data_in, float *fir_coeffs, float *data_downsampled_out, float osc_frequency, int chunk_number, float *debug_data_real, float *debug_data_imag)
{
    __shared__ float fir_coeffs_shared[Fir_length];
    __shared__ float mixed_data_re[Fir_length + SAMPLES_PER_BLOCK];
    __shared__ float mixed_data_im[Fir_length + SAMPLES_PER_BLOCK];

    //Stage 1: Load Data From memory, mix and store in shared memory
    //1.1 Load FIR coeffs into shared memory
    fir_coeffs_shared[threadIdx.x] = fir_coeffs[threadIdx.x];

    __syncthreads();

    // int inOffset = (blockIdx.x+1)*4096; //The plus 1 is because we skip the first block of data: assuming that it is wasted as we cannot access past values
    int inOffset = (blockIdx.x)*(SAMPLES_PER_BLOCK); //The plus 1 is because we skip the first block of data: assuming that it is wasted as we cannot access past values

    float sample_in = 0;
    int dgb_addr_dst = 0;

    int numLoads = SAMPLES_PER_BLOCK/blockDim.x; // Number of times each thread needs to read input data from global memory. Throws errors whem SAMPLES_PER_BLOCK != 4096 which needs to be investigated
    int numPrevLoads = Fir_length/blockDim.x;//This works out how many samples from the previous block need to be loaded taking into account the FIR size
    if (numPrevLoads == 0)
        numPrevLoads = 1;

    for(int k = -numPrevLoads; k < numLoads; k++){ //The -numPrevLoads accounts for the additional FIR length worth of data that we need to load of past values
        // 1.2 Load Data From Global Memory
        int index_in = inOffset + threadIdx.x + k*Fir_length;
  
        dgb_addr_dst = index_in + 256;

        if (index_in < 0)
        {
            sample_in = 0;
            // printf("block=%d and thread=%d with k=%d and index_in=%d and inoffset=%d with dgb_add= %d\n",blockIdx.x, threadIdx.x, k, index_in, inOffset,dgb_addr_dst);
        }else{
            // printf("block=%d and thread=%d with k=%d and index_in=%d and inoffset=%d with dgb_add= %d\n",blockIdx.x, threadIdx.x, k, index_in, inOffset,dgb_addr_dst);
            sample_in = data_in[index_in];
        }

        
        // The lookup index needs a Fir_length offset as the vector it will be multiplied with is later offset by this amount when appended to the 'mixed_data_re' vector.
        // int lookup_index = index_in + Fir_length;
        int lookup_index = index_in + Fir_length + N*(chunk_number);

        int mix_addr_dst = threadIdx.x + k * Fir_length + 256;

        //1.3 Mix value down
        float mixerValue_re;
        float mixerValue_im;
        float mixedSample_re;
        float mixedSample_im;
        float samples_per_cycle;
        float cycles;
        float lookup_step_size;

        samples_per_cycle = (float) (Fs / osc_frequency);

        lookup_step_size = (float) (1/samples_per_cycle)*2;

        float mixer_angle =  -1 * (lookup_index*lookup_step_size);

        // printf("Lookup\n");
        sincospif(mixer_angle, &mixerValue_im, &mixerValue_re);
        mixedSample_re = mixerValue_re * sample_in;
        mixedSample_im = mixerValue_im * sample_in;

        // debug_data_real[dgb_addr_dst] = sample_in;
        // debug_data_imag[dgb_addr_dst] = sample_in;
        // if (blockIdx.x == 0)
        // {
        //     debug_data_real[mix_addr_dst] = mixedSample_re;
        //     debug_data_imag[mix_addr_dst] = mixedSample_im;
        // }

        // debug_data_real[mix_addr_dst] = mixerValue_re;
        // debug_data_imag[mix_addr_dst] = mixerValue_im;

        //1.4 Store in memory. Offset by Fir_length as the first Fir_length samples will be fromthe held back slice from the previous
        mixed_data_re[mix_addr_dst] = mixedSample_re;
        mixed_data_im[mix_addr_dst] = mixedSample_im;

        // debug_data_real[dgb_addr_dst] = mixedSample_re;
        // debug_data_imag[dgb_addr_dst] = mixedSample_im;

        //# Debug
        // mixed_data_re[mix_addr_dst] = mixerValue_re;
        // mixed_data_im[mix_addr_dst] = mixerValue_im; 
        // mixed_data_re[mix_addr_dst] = sample_in;
        // mixed_data_im[mix_addr_dst] = sample_in;
    }

    // 2. Data has been mixed, now the fir will be applied. Needs to be synced before this happends
    __syncthreads();

    // 3. Mix Data and store in shared mixed_data array
    float sample_out_re = 0;
    float sample_out_im = 0;

    // int base_shared_mixed_sample_index = threadIdx.x * S + inOffset; 
    int base_shared_mixed_sample_index = threadIdx.x * 16; 
    
    int data_idx = 0;
    // 3.1 This for loop is where most of the kernels computation happens. Each iteration of the loop acceses a different location in shared memory. 
    // The kernel is heavily bound by the shared memory bandwidth. One isse is that there are many bank conflicts in the shared memory access in 
    // this loop. Thread 0 acceses element 0. thread 2 acceses element 16, thread 3 acceses element 32, etc. This is almost the worst case shared memory
    // access pattern. By improving this access pattern it may be possible to get a significant improvement in the kernel performance. (I want to say 10x
    // improvement but its difficult to be 100% sure of this.).
   
    for(int i = 0; i < Fir_length; i++){
        data_idx = base_shared_mixed_sample_index - i + (Fir_length-1); // The 255 is added to offset the address so the adressing is flipped (as required for convolution)
        // int dbg_idx = data_idx + (blockIdx.x) * N;

        float fir_coeff = fir_coeffs_shared[i];
        float mixedSample_re = mixed_data_re[data_idx];
        float mixedSample_im = mixed_data_im[data_idx];

        // if (blockIdx.x == 0)
        // {
        //     debug_data_real[data_idx] = mixedSample_re;
        //     debug_data_imag[data_idx] = mixedSample_im;
        // }

        sample_out_re = sample_out_re + mixedSample_re * fir_coeff;
        sample_out_im = sample_out_im + mixedSample_im * fir_coeff;
        
    }

    int index_out = (blockIdx.x*blockDim.x + threadIdx.x)*2;
    int dbg_idx = (blockIdx.x*blockDim.x + threadIdx.x);

    // debug_data_real[dbg_idx] = sample_out_re;
    // debug_data_imag[dbg_idx] = sample_out_im;

    data_downsampled_out[index_out] = sample_out_re;
    data_downsampled_out[index_out+1] = sample_out_im;
    // data_downsampled_out[index_out] = 0;
    // data_downsampled_out[index_out+1] = 0;   
    
}
