#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <mma.h>

extern "C" {  //See the .py file for an explanation of why this is necessary.
    
__global__ void simple_tc_matmul(__half A[TCM_SIZE][TCM_SIZE], __half B[TCM_SIZE][TCM_SIZE], float C[TCM_SIZE][TCM_SIZE])
{
    using namespace nvcuda::wmma;

    fragment<matrix_a, TCM_SIZE, TCM_SIZE, TCM_SIZE, half, row_major> a_frag;
    fragment<matrix_b, TCM_SIZE, TCM_SIZE, TCM_SIZE, half, row_major> b_frag;
    fragment<accumulator, TCM_SIZE, TCM_SIZE, TCM_SIZE, float> c_frag;

    fill_fragment(c_frag, 0.0f);

    load_matrix_sync(a_frag, A[0], TCM_SIZE);
    load_matrix_sync(b_frag, B[0], TCM_SIZE);
    mma_sync(c_frag, a_frag, b_frag, c_frag);
    store_matrix_sync(C[0], c_frag, TCM_SIZE, mem_row_major);
} 

}