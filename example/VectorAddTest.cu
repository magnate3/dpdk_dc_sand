#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "VectorAddTest.hpp"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/// The constructor adds a parameter which assigns the size of the vectors.
VectorAddTest::VectorAddTest(size_t uVectorLength) : m_uVectorLength(uVectorLength)
{
    /// Turns out that using pinned host memory is MUCH faster than the non-pinned variety.
    /// This sped things up by a factor of about 3 on my tests on qgpu02.
    hipHostMalloc((void**)&m_piHVectorA, sizeof(*m_piHVectorA)*m_uVectorLength);
    hipHostMalloc((void**)&m_piHVectorB, sizeof(*m_piHVectorB)*m_uVectorLength);
    hipHostMalloc((void**)&m_piHVectorC, sizeof(*m_piHVectorC)*m_uVectorLength);

    gpuErrchk(hipMalloc((void **) &m_piDVectorA, m_uVectorLength*sizeof(*m_piDVectorA)));
    gpuErrchk(hipMalloc((void **) &m_piDVectorB, m_uVectorLength*sizeof(*m_piDVectorB)));
    gpuErrchk(hipMalloc((void **) &m_piDVectorC, m_uVectorLength*sizeof(*m_piDVectorC)));
}


/// The destructor just cleans up.
VectorAddTest::~VectorAddTest()
{
    hipFree(m_piHVectorA);
    hipFree(m_piHVectorB);
    hipFree(m_piHVectorC);

    hipFree(m_piDVectorA);
    hipFree(m_piDVectorB);
    hipFree(m_piDVectorC);
}


/// Simulated input data in this case is a ramp up on one vector, a ramp down on the other. Each element of the sum should be the same.
void VectorAddTest::simulate_input()
{
    for (size_t i = 0; i < m_uVectorLength; i++)
    {
        m_piHVectorA[i] = i;
        m_piHVectorB[i] = m_uVectorLength - i;
    }
}


/// Simple transfer to device memory.
void VectorAddTest::transfer_HtoD()
{
    gpuErrchk(hipMemcpy(m_piDVectorA, m_piHVectorA, m_uVectorLength*sizeof(*m_piHVectorA), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(m_piDVectorB, m_piHVectorB, m_uVectorLength*sizeof(*m_piHVectorA), hipMemcpyHostToDevice));
}


//Kernel adds A and B, storing the result to C.
__global__ void kernel_vector_add(int *A, int *B, int *C, size_t N)
{
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < N) //in case the size of the operation doesn't fit neatly into block size.
    {
        //for (int i = 0; i < 1000; i++) // To make the GPU spin for a while
            C[tid] = A[tid] + B[tid];
    }
}


void VectorAddTest::run_kernel()
{
    //Work out some kind of dimensionality for the kernel.
    //This example is nearly trivial since it needs virtually no memory, but this can often be
    //fairly critical for good utilisation of the GPU.
    int blockSize = 256;
    int numBlocks = (m_uVectorLength + blockSize - 1) / blockSize;
    kernel_vector_add<<< numBlocks, blockSize >>>(m_piDVectorA, m_piDVectorB, m_piDVectorC, m_uVectorLength);
    gpuErrchk(hipGetLastError());
}


void VectorAddTest::transfer_DtoH()
{
    gpuErrchk(hipMemcpy(m_piHVectorC, m_piDVectorC, m_uVectorLength*sizeof(*m_piHVectorC), hipMemcpyDeviceToHost));
}


void VectorAddTest::verify_output()
{
    for (size_t i = 0; i < m_uVectorLength; i++)
    {
        if (m_piHVectorC[i] != (int) m_uVectorLength)
        {
            m_iResult = -1;
            std::cout << "Element " << i << " not equal. Expected " << m_uVectorLength << " but got " << m_piHVectorC[i] << "!\n";
            return;
        }
    }
    m_iResult = 1;
}